#include "hip/hip_runtime.h"
#include <auxf.h>

#define MULTIPLY 0
#define MULTIPLY_TRANSPOSE 1

__global__ void transpose(float *in, float *out, int m, int n)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

int arrayInIndex = j + i*n;
int arrayOutIndex = i + j*m;

out[arrayOutIndex] = in[arrayInIndex];
}

__global__ void multiply(float *a, float *b, float *c, int m, int n, int k)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

double sum = .0;
for (int s = 0; s < n; ++s) {
sum += a[i*n + s]*b[s*k + j];
}

c[i*k + j] = sum;
}

__global__ void multiply_tr(float *a, float *b, float *c, int m, int n, int k)
{
int i = threadIdx.x + blockIdx.x*blockDim.x;
int j = threadIdx.y + blockIdx.y*blockDim.y;

double sum = .0;
for (int s = 0; s < n; ++s) {
sum += a[i*n + s]*b[j*n + s];
}

c[i*k + j] = sum;
}

int main(int argc, char *argv[])
{
int mode = MULTIPLY;
if (argc > 1) {
mode = atoi(argv[1]);
}

float *a, *b, *c, *tb;
float *da, *db, *dc;
  
a = (float *)malloc(m*n*sizeof(float));
b = (float *)malloc(n*k*sizeof(float));
c = (float *)malloc(m*k*sizeof(float));
  
hipMalloc((void **)&da, m*n*sizeof(float));
hipMalloc((void **)&db, n*k*sizeof(float));
hipMalloc((void **)&dc, m*k*sizeof(float));
hipMalloc((void **)&tb, n*k*sizeof(float));
fillMatrixMult(a, b, m, n, k);
 
hipMemcpy(da, a, m*n*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(db, b, n*k*sizeof(float), hipMemcpyHostToDevice);

dim3 tDimGrid(n / BLOCK_SIZE, k / BLOCK_SIZE);
dim3 dimGrid(m / BLOCK_SIZE, k / BLOCK_SIZE);
dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

transpose<<<tDimGrid, dimBlock>>>(db, tb, n, k);
hipDeviceSynchronize();

hipEvent_t start, stop;
float elapsedTime;
hipEventCreate(&start);
hipEventCreate(&stop);
  
hipEventRecord(start, 0);
if (mode == MULTIPLY_TRANSPOSE) {
multiply_tr<<<dimGrid, dimBlock>>>(da, tb, dc, m, n, k);
} else {
multiply<<<dimGrid, dimBlock>>>(da, db, dc, m, n, k);
}
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

hipEventElapsedTime(&elapsedTime, start, stop);
hipMemcpy(c, dc, m*k*sizeof(float), hipMemcpyDeviceToHost);

printf("Elapsed: %g\n", elapsedTime);
//printMatrix(c, m, k);
checkResult(c, m, k);

return 0;
}