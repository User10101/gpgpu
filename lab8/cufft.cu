
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>

#define BLOCK_SIZE 128
#define NX (BLOCK_SIZE * 1)
#define BATCH 1
#define pi 3.141592

__global__ void gInitData(hipfftComplex *data, hiprandState *state)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  data[i].x = hiprand_uniform(&state[i])*8.15*cosf(2*pi*3*i/NX) + hiprand_uniform(&state[i])*6.75*sinf(2*pi*5*i/NX);
  data[i].y = 0.0f;
}

__global__ void init_stuff(hiprandState *state)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(1337, idx, 0, &state[idx]);
}

int main()
{
  hipfftHandle plan;
  hipfftComplex *data;
  hipfftComplex *data_h = (hipfftComplex *)calloc(NX, sizeof(hipfftComplex));

  hipMalloc((void **)&data, sizeof(hipfftComplex) * NX * BATCH);
  if (hipGetLastError() != hipSuccess) {
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return -1;
  }

  hiprandState *d_state;
  hipMalloc(&d_state, NX);
  init_stuff<<<(NX)/(BLOCK_SIZE), BLOCK_SIZE>>>(d_state);
  
  gInitData<<<(NX)/(BLOCK_SIZE), BLOCK_SIZE>>>(data, d_state);
  hipDeviceSynchronize();

  // cudaMemcpy(data_h, data, NX * BATCH * sizeof(cufftComplex), cudaMemcpyDeviceToHost);
  // for (size_t i = 0; i < NX; ++i) {
  //   printf("%f\t%f\n", data_h[i].x, data_h[i].y);
  // }
  // printf("\n");
  
  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return -1;
  }

  if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return -1;
  }

  if (hipDeviceSynchronize() != hipSuccess) {
      fprintf(stderr, "Cuda error: Failed to syncrhonize\n");
      return -1;
  }

  hipMemcpy(data_h, data, NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < NX/2; ++i) {
    printf("%d\t%f\n", i, sqrt(pow(data_h[i].x, 2) + pow(data_h[i].y, 2)));
  }

  hipfftDestroy(plan);
  hipFree(d_state);
  hipFree(data);
  free(data_h);

  return 0;
}
