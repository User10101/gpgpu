
#include <hip/hip_runtime.h>
#include <iostream>

#define N 100000

int main(int argc, char *argv[])
{
  bool async_mode = false;
  if (argc > 1) {
    async_mode = true;
  }
  float *adev;
  hipMalloc((void **)&adev, N * sizeof(float));

  if (!async_mode) {
    float *a = (float *)malloc(N * sizeof(float));
    hipMemcpy(adev, a, N * sizeof(float), hipMemcpyHostToDevice);
    free(a);
  } else {
    hipStream_t stream;
    hipStreamCreate(&stream);
    float *ap;
    hipHostAlloc((void **)&ap, N * sizeof(float), hipHostMallocDefault);
    hipMemcpyAsync(adev, ap, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    hipHostFree(ap);
  }
  hipFree(adev);
}