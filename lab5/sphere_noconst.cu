#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI 3.14159265358979323846
#define COEF 48
#define VERTCOUNT COEF*COEF*2//-(COEF-1)*2
#define RADIUS 10.0f
#define FGSIZE 20
#define FGSHIFT FGSIZE/2
#define IMIN(A,B) (A<B?A:B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID IMIN(32,(VERTCOUNT+THREADSPERBLOCK-1)/THREADSPERBLOCK)

typedef float(*ptr_f)(float, float, float);

struct Vertex
{
  float x, y, z;
};

texture<float, 3, hipReadModeElementType> df_tex;
hipArray* df_Array = 0;

__global__ void kernel(float *a, Vertex *vert)
{
  __shared__ float cache[THREADSPERBLOCK];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
	
  float x = vert[tid].x + FGSHIFT + 0.5f;
  float y = vert[tid].y + FGSHIFT + 0.5f;
  float z = vert[tid].z + FGSHIFT + 0.5f;
  cache[cacheIndex] = tex3D(df_tex, z, y, x);

  __syncthreads();
  for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
      if (cacheIndex < s)
	cache[cacheIndex] += cache[cacheIndex + s];
      __syncthreads();
    }

  if (cacheIndex == 0)
    a[blockIdx.x] = cache[0];
}	

float func(float x, float y, float z)
{
  return (0.5*sqrtf(15.0/M_PI))*(0.5*sqrtf(15.0/M_PI))*z*z*y*y*sqrtf(1.0f-z*z/RADIUS/RADIUS)/RADIUS/RADIUS/RADIUS/RADIUS;
}

void calc_f(float *arr_f, int x_size, int y_size, int z_size, ptr_f f)
{
  for (int x = 0; x < x_size; ++x)
    for (int y = 0; y < y_size; ++y)
      for (int z = 0; z < z_size; ++z)
	arr_f[z_size * (x * y_size + y) + z] = f(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
}

float check(Vertex *v, ptr_f f)
{
  float sum = 0.0f;
  for (int i = 0; i < VERTCOUNT; ++i)
    sum += f(v[i].x, v[i].y, v[i].z);
		
  return sum;
}

void init_vertexes(Vertex *dev_vert)
{
  Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERTCOUNT);
  int i = 0;
  for (int iphi = 0; iphi < 2 * COEF; ++iphi)
    {	
      for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i)
	{
	  float phi = iphi * M_PI / COEF;
	  float psi = ipsi * M_PI / COEF;
	  temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
	  temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
	  temp_vert[i].z = RADIUS * cosf(psi);
	}
    }
  printf("sumcheck = %f\n", check(temp_vert, &func)*M_PI*M_PI/ COEF/COEF);
  hipMemcpy(dev_vert, temp_vert, VERTCOUNT * sizeof(Vertex), hipMemcpyHostToDevice);
  free(temp_vert);
}

void init_texture(float *df_h)
{
  const hipExtent volumeSize = make_hipExtent(FGSIZE, FGSIZE, FGSIZE);
  hipChannelFormatDesc  channelDesc=hipCreateChannelDesc<float>();
  hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
  hipMemcpy3DParms  cpyParams={0};
  cpyParams.srcPtr = make_hipPitchedPtr( (void*)df_h, volumeSize.width*sizeof(float),  volumeSize.width,  volumeSize.height);
  cpyParams.dstArray = df_Array;
  cpyParams.extent = volumeSize;
  cpyParams.kind = hipMemcpyHostToDevice; 
  hipMemcpy3D(&cpyParams);
  df_tex.normalized = false;
  df_tex.filterMode = hipFilterModeLinear;
  df_tex.addressMode[0] = hipAddressModeClamp;
  df_tex.addressMode[1] = hipAddressModeClamp;
  df_tex.addressMode[2] = hipAddressModeClamp;
  hipBindTextureToArray(df_tex, df_Array, channelDesc);
}

void release_texture()
{
  hipUnbindTexture(df_tex); 
  hipFreeArray(df_Array);
}

int main(void)
{
  Vertex *vert;
  hipMalloc((void **)&vert, VERTCOUNT * sizeof(Vertex));
  init_vertexes(vert);

  float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
  calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);
  init_texture(arr);

  float *sum = (float*)malloc(sizeof(float) * BLOCKSPERGRID);
  float *sum_dev;
  hipMalloc((void**)&sum_dev, sizeof(float) * BLOCKSPERGRID);	

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);	

  kernel<<<BLOCKSPERGRID,THREADSPERBLOCK>>>(sum_dev, vert);

  hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost);
  float s = 0.0f;
  for (int i = 0; i < BLOCKSPERGRID; ++i)
    s += sum[i];
  printf("sum = %f\n", s*M_PI*M_PI / COEF/COEF);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("Time: %3.1f ms\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(sum_dev);
  hipFree(vert);
  free(sum);
  release_texture();
  free(arr);

  return 0;
}