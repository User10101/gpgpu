#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>

__global__ void gInit(float *a, float *b, int N)
{
  int thread_id = blockIdx.x + blockDim.x*threadIdx.x;

  unsigned int seed = thread_id;
  hiprandState s;
  hiprand_init(seed, 0, 0, &s);
  
  for (int i = thread_id; i < N; i += blockDim.x*gridDim.x) {
    a[i] = hiprand_uniform(&s);
    b[i] = hiprand_uniform(&s);
  }
}

__global__ void gSum(float *a, float *b, float *c, int N)
{
  int thread_id = blockIdx.x + blockDim.x*threadIdx.x;
  for (int i = thread_id; i < N; i += blockDim.x*gridDim.x) {
    c[i] = a[i] + b[i];
  }
}

int main(int argc, char *argv[])
{
  float *a, *b, *c;
  float *ha, *hb, *hc;
  if (argc < 4) {
    fprintf(stderr, "USAGE:prog <blocks> <threads> <offset>\n");
    return 1;
  }

  int num_of_blocks = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);
  int N = num_of_blocks*threads_per_block;
  int offset = atoi(argv[3]);

  hipMalloc((void **)&a, (N + offset)*sizeof(float));
  hipMalloc((void **)&b, N*sizeof(float));
  hipMalloc((void **)&c, N*sizeof(float));

  ha = (float *)calloc(N, sizeof(float));
  hb = (float *)calloc(N, sizeof(float));
  hc = (float *)calloc(N, sizeof(float));

  gInit<<<num_of_blocks, threads_per_block>>>(a + offset, b, N);
  hipDeviceSynchronize();

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  gSum<<<num_of_blocks, threads_per_block>>>(a + offset, b, c, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  fprintf(stderr, "gTest took %g\n", elapsedTime);

  hipMemcpy(ha, a + offset, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hb, b, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hc, c, N*sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < N; ++i) {
  //   printf("%g\t%g\t%g\n", ha[i], hb[i], hc[i]);
  // }

  hipFree(a);
  hipFree(b);
  hipFree(c);
  free(ha);
  free(hb);
  free(hc);

  return 0;
}