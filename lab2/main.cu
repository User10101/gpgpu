#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>

__global__ void gInit(float *a, float *b, int N, int offset)
{
  int thread_id = threadIdx.x + blockDim.x*blockIdx.x + offset;

  unsigned int seed = thread_id;
  hiprandState s;
  hiprand_init(seed, 0, 0, &s);
  
  for (int i = thread_id; i < N + offset; i += blockDim.x*gridDim.x) {
    a[i] = hiprand_uniform(&s);
    b[i] = hiprand_uniform(&s);
  }
}

__global__ void gSum(float *a, float *b, float *c, int N, int offset)
{
  int thread_id = threadIdx.x + blockIdx.x*blockDim.x + offset;
  for (int i = thread_id; i < N + offset; i += blockDim.x*gridDim.x) {
    c[i] = a[i] + b[i];
  }
}

int main(int argc, char *argv[])
{
  float *a, *b, *c;
  float *ha, *hb, *hc;
  if (argc < 4) {
    fprintf(stderr, "USAGE:prog <blocks> <threads> <offset>\n");
    return 1;
  }

  int num_of_blocks = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);
  int N = num_of_blocks*threads_per_block;
  int offset = atoi(argv[3]);

  hipMalloc((void **)&a, (N + offset)*sizeof(float));
  hipMalloc((void **)&b, (N + offset)*sizeof(float));
  hipMalloc((void **)&c, (N + offset)*sizeof(float));

  ha = (float *)calloc(N + offset, sizeof(float));
  hb = (float *)calloc(N + offset, sizeof(float));
  hc = (float *)calloc(N + offset, sizeof(float));

  float cumTime = .0;
  int n_opts = 10;
  for (int i = 0; i < n_opts; ++i) {
    gInit<<<num_of_blocks, threads_per_block>>>(a, b, N, offset);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gSum<<<num_of_blocks, threads_per_block>>>(a, b, c, N, offset);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cumTime += elapsedTime;
  }
  printf("%g\n", cumTime / n_opts);

  hipMemcpy(ha, a, (N + offset)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hb, b, (N + offset)*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hc, c, (N + offset)*sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < N; ++i) {
  //   printf("%g\t%g\t%g\n", ha[i], hb[i], hc[i]);
  // }

  hipFree(a);
  hipFree(b);
  hipFree(c);
  free(ha);
  free(hb);
  free(hc);

  return 0;
}