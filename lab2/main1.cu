#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>
#include <math.h>

struct vec3d
{
  float x;
  float y;
  float z;
};

__global__ void strInit(struct vec3d *a)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  a[i].x = i;
  a[i].y = i / 2;
  a[i].z = i + 1;
}

__global__ void strLength(struct vec3d *v, float *length)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  length[i] = sqrtf(v[i].x*v[i].x + v[i].y*v[i].y + v[i].z*v[i].z);
}

int main(int argc, char *argv[])
{
  struct vec3d *v;
  struct vec3d *hv;
  float *length, *hlength;
  
  if (argc < 3) {
    fprintf(stderr, "USAGE:prog <blocks> <threads>\n");
    return 1;
  }

  int num_of_blocks = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);
  int N = num_of_blocks*threads_per_block;

  hipMalloc((void **)&v, N*sizeof(struct vec3d));
  hipMalloc((void **)&length, N*sizeof(float));
  hv = (struct vec3d *)calloc(N, sizeof(struct vec3d));
  hlength = (float *)calloc(N, sizeof(float));
  
  strInit<<<num_of_blocks, threads_per_block>>>(v);
  hipDeviceSynchronize();

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  strLength<<<num_of_blocks, threads_per_block>>>(v, length);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  fprintf(stderr, "gTest took %g\n", elapsedTime);

  hipMemcpy(hlength, length, N*sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < N; ++i) {
  //   printf("%g\t%g\t%g\n", ha[i], hb[i], hc[i]);
  // }

  hipFree(v);
  hipFree(length);
  free(hv);
  free(hlength);

  return 0;
}
