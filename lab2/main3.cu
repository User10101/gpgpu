#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <malloc.h>

__global__ void gInit(float *a, int N, int stride)
{
  int thread_id = threadIdx.x + blockDim.x*blockIdx.x;

  unsigned int seed = thread_id;
  hiprandState s;
  hiprand_init(seed, 0, 0, &s);
  
  for (int i = thread_id; i < N; i += blockDim.x*gridDim.x) {
    a[i] = hiprand_uniform(&s);
  }
}

__global__ void gCpy(float *a, float *b, int N, int stride)
{
  int thread_id = (threadIdx.x + blockIdx.x*blockDim.x)*stride;
  for (int i = thread_id; i < N; i += blockDim.x*gridDim.x) {
    a[i] = b[i];
  }
}

int main(int argc, char *argv[])
{
  float *a, *b;
  float *ha, *hb;
  if (argc < 4) {
    fprintf(stderr, "USAGE:prog <blocks> <threads> <stride>\n");
    return 1;
  }

  int num_of_blocks = atoi(argv[1]);
  int threads_per_block = atoi(argv[2]);
  int N = num_of_blocks*threads_per_block;
  int stride = atoi(argv[3]);

  hipMalloc((void **)&a, N*sizeof(float));
  hipMalloc((void **)&b, N*sizeof(float));

  ha = (float *)calloc(N, sizeof(float));
  hb = (float *)calloc(N, sizeof(float));

  float cumTime = .0;
  int n_opts = 10;
  for (int i = 0; i < n_opts; ++i) {
    gInit<<<num_of_blocks, threads_per_block>>>(a, N, stride);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gCpy<<<num_of_blocks, threads_per_block>>>(a, b, N, stride);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cumTime += elapsedTime;
  }
  printf("%g\n", cumTime / n_opts);

  hipMemcpy(ha, a, N*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hb, b, N*sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < N; ++i) {
  //   printf("%g\t%g\t%g\n", ha[i], hb[i], hc[i]);
  // }

  hipFree(a);
  hipFree(b);
  free(ha);
  free(hb);

  return 0;
}